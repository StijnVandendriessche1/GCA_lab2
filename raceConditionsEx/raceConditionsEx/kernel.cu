#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

#include <chrono>

// ---- reduction GPU
// -------- GPU recution kernel
__global__ void getMaxReduction(int* A, int* max, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = N;
    for (int n = 0; n < (log2f(N)); n++)
    {
        j = j / 2;
        if (i < j)
        {
            if (A[i] < A[i + j])
            {
                A[i] = A[i + j];
            }
        }
        __syncthreads();
    }
    if (i == 0)
    {
        *max = A[0];
    }
}

//-------- GPU reduction main function
int detectMaxGPUReduction(int* A, int N)
{
    //int for maximum
    int max = 0;

    // Allocate the device input vector
    int* gpuA = NULL;
    hipMalloc((void**)&gpuA, N * sizeof(int));

    // Allocate the device output int (where the max will be stored)
    int* gpuMax = NULL;
    hipMalloc((void**)&gpuMax, sizeof(int));

    // Copy the host input vector nd output int in host memory to the device input
    hipMemcpy(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuMax, &max, sizeof(int), hipMemcpyHostToDevice);

    //execute kernel
    int threadsPerBlock = 1024;
    //int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid = 1;
    getMaxReduction << <blocksPerGrid, threadsPerBlock >> > (gpuA, gpuMax, N);

    //copy result back to host
    hipMemcpy(&max, gpuMax, sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(A, gpuA, N * sizeof(int), hipMemcpyDeviceToHost);

    //free up memory from GPU
    hipFree(gpuA);
    hipFree(gpuMax);

    //max = A[0];
    return max;
}


// ---- GPU atomic
// -------- GPU atomic kernel
__global__ void getMax(int* A, int* max)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    atomicMax(max, A[i]);
}
// -------- GPU atomic main function
int detectMaxGPU(int* A, int N)
{   
    //int for maximum
    int max = 0;

    // Allocate the device input vector
    int* gpuA = NULL;
    hipMalloc((void**)&gpuA, N * sizeof(int));

    // Allocate the device output int (where the max will be stored)
    int* gpuMax = NULL;
    hipMalloc((void**)&gpuMax, sizeof(int));

    // Copy the host input vector nd output int in host memory to the device input
    hipMemcpy(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuMax, &max, sizeof(int), hipMemcpyHostToDevice);

    //execute kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    getMax << <blocksPerGrid, threadsPerBlock >> > (gpuA, gpuMax);

    //copy result back to host
    hipMemcpy(&max, gpuMax, sizeof(int), hipMemcpyDeviceToHost);

    //free up memory from GPU
    hipFree(gpuA);
    hipFree(gpuMax);

    return max;
}

// ---- CPU
// -------- CPU function
int detectMaxCPU(int* A, int size)
{
    //cpu algorithm
    int max = INT_MIN;
    for (int i = 0; i < size; i++)
    {
        if (A[i] > max)
        {
            max = A[i];
        }
    }
    return max;
}

//main function
int main()
{
    const int base = 2;
    for (int i = 0; i < 11; i++)
    {
        //create a random array of integers
        //allocate memory
        int N = (int)base * pow(2, i); //mount of elements
        size_t size = N * sizeof(int); //amount of bytes
        int* A = (int*)malloc(size); //memory alocation

        //fill up the memory with a random array
        for (int i = 0; i < N; ++i) {
            A[i] = rand();
        }

        //three executions types
        //execute on cpu
        printf("maximum of %d numbers CPU: %d\n\n", N, detectMaxCPU(A, N));

        //execute on GPU
        printf("maximum of %d numbers GPU: %d\n\n", N, detectMaxGPU(A, N));

        //execute on GPU using reduction
        printf("maximum of %d numbers GPU: %d\n\n - - - - \n", N, detectMaxGPUReduction(A, N));

        //free up memory
        free(A);
    }
    return 0;
}
